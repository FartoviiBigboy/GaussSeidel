#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <string>
#include <cmath>
#include <omp.h>
#include <string>

const int threadsPerBlock = 256;
int blocksPerGrid;

using namespace std;



int imin(int a, int b) {
	return (a < b ? a : b);
}


double getRandomNumber(double min, double max)
{
	static const double fraction = 1.0 / ((double)(RAND_MAX)+1.0);
	return rand() * fraction * (max - min + 1.0) + min;
}




__global__ void tempSum(const double* matrixA, const double* vectorB, double* vectorX, double* vectorSum, int* N, int* numberOfIteration) {

	int tempNumberOfIteration = (*numberOfIteration);

	int tid;

	tid = threadIdx.x + blockIdx.x * blockDim.x;

	while (tid < (*N)) {

		if (tempNumberOfIteration == 0) {
			vectorSum[tid] = vectorX[tid];
			vectorX[tid] = vectorB[tid] / matrixA[tid * (*N) + tid];

		}

		if (tempNumberOfIteration == 1) {
			for (int i = tid + 1; i < (*N); i++) {
				vectorX[tid] -= vectorSum[i] * matrixA[tid * (*N) + i] / matrixA[tid * (*N) + tid];

			}

		}

		if ((tid > (tempNumberOfIteration - 2)) && (tempNumberOfIteration > 1)) {
			vectorX[tid] -= vectorX[tempNumberOfIteration - 2] * matrixA[tid * (*N) + tempNumberOfIteration - 2] / matrixA[tid * (*N) + tid];
		}


		tid += blockDim.x * gridDim.x;
	}

	tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid == 0) {
		if ((*numberOfIteration) != (*N)) {
			(*numberOfIteration) = (*numberOfIteration) + 1;
		}
		else {
			(*numberOfIteration) = 0;
		}
	}

}





int main(int argc, char* argv[])
{

	srand(time(NULL));

	ifstream input;

	string fileName;

	int menuKey;

	int N;
	int* d_N;

	double* matrixA;
	double* vectorB;
	double* vectorX;
	double* vectorSum;

	double* d_matrixA;
	double* d_vectorB;
	double* d_vectorX;
	double* d_vectorSum;

	int numberOfIteration;
	int* d_numberOfIteration;

	double Eps;

	double time1;
	double time2;

	double* multipleAnswer;


	cout << "Choose epsilon:" << endl;
	cin >> Eps;

	cout << "Choose mode for loading data: " << endl << "1 - generate" << endl << "2 - from file" << endl;
	cin >> menuKey;

	if (menuKey == 1) {
		double min;
		double max;
		cout << "Enter N: " << endl;
		cin >> N;
		cout << "Enter min and max separated by space: " << endl;
		cin >> min >> max;

		matrixA = new double[N * N];
		vectorB = new double[N];
		vectorX = new double[N];
		vectorSum = new double[N];


		for (int i = 0; i < N; i++) {
			for (int j = 0; j < N + 1; j++) {
				if (j == N) {
					vectorB[i] = getRandomNumber(min, max);
				}
				else {
					matrixA[i * N + j] = getRandomNumber(min, max);
				}
			}
		}

		double megaSum = 0;

		for (int i = 0; i < N; i++) {
			for (int j = 0; j < N; j++) {
				if (i != j) {
					megaSum += fabs(matrixA[i * N + j]);
				}
			}
		}

		for (int i = 0; i < N; i++) {
			matrixA[i * N + i] < 0 ? matrixA[i * N + i] -= megaSum * 1 : matrixA[i * N + i] += megaSum * 1;
		}

	}
	if (menuKey == 2) {
		cin.get();
		cout << "Choose file name for matrix:" << endl;
		getline(cin, fileName);


		input.open(fileName.c_str(), ios_base::in);
		input >> N;

		matrixA = new double[N * N];
		vectorB = new double[N];
		vectorX = new double[N];
		vectorSum = new double[N];

		for (int i = 0; i < N; i++) {
			for (int j = 0; j < N + 1; j++) {
				if (j == N) {
					input >> vectorB[i];
				}
				else {
					input >> matrixA[i * N + j];
				}
			}
		}
		input.close();
	}



	for (int i = 0; i < N; i++) {
		vectorX[i] = 0;
		vectorSum[i] = 0;
	}

	numberOfIteration = 0;

	blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);



	hipMalloc((void**)&d_matrixA, N * N * sizeof(double));
	hipMalloc((void**)&d_vectorB, N * sizeof(double));
	hipMalloc((void**)&d_vectorX, N * sizeof(double));
	hipMalloc((void**)&d_vectorSum, N * sizeof(double));

	hipMalloc((void**)&d_N, sizeof(int));

	hipMalloc((void**)&d_numberOfIteration, sizeof(int));


	hipMemcpy(d_matrixA, matrixA, N * N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_vectorB, vectorB, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_vectorX, vectorX, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_vectorSum, vectorSum, N * sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(d_N, &N, sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(d_numberOfIteration, &numberOfIteration, sizeof(int), hipMemcpyHostToDevice);


	bool check = true;

	int countOfOperation = 0;

	time1 = omp_get_wtime();

	while ((check != false) && (countOfOperation <= 100)) {
		check = false;
		for (int i = 0; i < N + 1; i++) {
			tempSum << <blocksPerGrid, threadsPerBlock >> > (d_matrixA, d_vectorB, d_vectorX, d_vectorSum, d_N, d_numberOfIteration);
		}
		hipMemcpy(vectorX, d_vectorX, N * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(vectorSum, d_vectorSum, N * sizeof(double), hipMemcpyDeviceToHost);


		countOfOperation++;
		for (int i = 0; i < N; i++) {
			if (fabs(vectorX[i] - vectorSum[i]) > Eps) {
				check = true;
				break;
			}
		}
		if (check == true) {
			continue;
		}
		else {
			break;
		}


	}

	time1 = omp_get_wtime() - time1;

	multipleAnswer = new double[N];

	for (int i = 0; i < N; i++) {
		multipleAnswer[i] = vectorX[i];
	}


	hipFree(d_matrixA);
	hipFree(d_vectorB);
	hipFree(d_vectorX);

	hipFree(d_N);
	hipFree(d_numberOfIteration);

	cout << countOfOperation << endl;

	for (int i = 0; i < N; i++) {
		vectorX[i] = 0;
	}

	double* tempVec = new double[N];
	for (int i = 0; i < N; i++) {
		tempVec[i] = 0;
	}



	check = true;

	countOfOperation = 0;

	time2 = omp_get_wtime();

	while ((check != false) && (countOfOperation <= 100)) {
		check = false;

		for (int j = 0; j < N; j++) {
			vectorSum[j] = vectorX[j];
			vectorX[j] = vectorB[j] / matrixA[j * N + j];
			for (int k = 0; k < N; k++) {
				if (k == j) {
					continue;
				}
				vectorX[j] = vectorX[j] - ((matrixA[j * N + k] / matrixA[j * N + j]) * tempVec[k]);
				tempVec[j] = vectorX[j];
			}
		}

		countOfOperation++;
		for (int i = 0; i < N; i++) {
			if (fabs(vectorX[i] - vectorSum[i]) > Eps) {
				check = true;
				break;
			}
		}
		if (check == true) {
			continue;
		}
		else {
			break;
		}


	}

	time2 = omp_get_wtime() - time2;

	cout << countOfOperation << endl;

	cout << endl;

	check = false;

	for (int i = 0; i < N; i++) {
		if (fabs(vectorX[i] - multipleAnswer[i]) > Eps) {
			check = true;
			break;
		}
	}

	if (!check) {
		cout << "Values are equal!" << endl;
	}
	else {
		cout << "Values are not equal!" << endl;
	}

	/*for (int j = 0; j < N; j += 100) {
		double sum = 0;
		for (int i = 0; i < N; i++) {
			sum += matrixA[j * N + i] * vectorX[i];
		}
		cout << sum << " " << vectorB[j] << endl;
	}*/


	cout << "Time for multiple: " << time1 << endl;
	cout << "Time for single: " << time2 << endl;


	ofstream output;

	cout << "Do you want to save CLAY?: " << endl << "1 - yes" << endl << "2 - no" << endl;
	cin >> menuKey;
	if (menuKey == 1) {
		cout << "Choose file name for CLAY: " << endl;
		cin.get();
		getline(cin, fileName);
		output.open(fileName.c_str(), ios_base::out);
		output << N << endl;
		for (int i = 0; i < N; i++) {
			for (int j = 0; j < N + 1; j++) {
				if (j == N) {
					output << vectorB[i] << endl;
				}
				else {
					output << matrixA[i * N + j] << " ";
				}
			}
		}
		output.close();


	}
	cout << "Do you want to save multiple and single answers: " << endl << "1 - yes" << endl << "2 - no" << endl;
	cin >> menuKey;
	if (menuKey == 1) {
		cout << "Choose file name for multiple: " << endl;
		cin.get();
		getline(cin, fileName);
		output.open(fileName.c_str(), ios_base::out);
		output << N << endl;
		for (int i = 0; i < N; i++) {
			output << multipleAnswer[i] << endl;
		}
		output.close();

		cout << "Choose file name for single: " << endl;
		getline(cin, fileName);
		output.open(fileName.c_str(), ios_base::out);
		output << N << endl;
		for (int i = 0; i < N; i++) {
			output << vectorX[i] << endl;
		}
		output.close();
	}

	return 0;
}
